#include "cfd.h"
#include <cmath>

int main(){
	hipSetDevice(1);
	hipDeviceSynchronize();
	CFD cfd = CFD(1000, 10, 10);
	double maxTime = 1;
	cfd.step(maxTime);
	return 0;
}