#include "cfd.h"
#include <cmath>

int main(){
	hipSetDevice(1);
	hipDeviceSynchronize();
	CFD cfd = CFD(10000, 10, 10);
	double maxTime = 1;
	cfd.step(maxTime);
	return 0;
}