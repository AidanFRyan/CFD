#include "hip/hip_runtime.h"
#include "cfd.h"


__global__ void initialize(double* a, double* oA, double* x, double totalSize, int n, int ghosts){
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	for(int j = 0; blockDim.x*j + i < n; j++){
		int index = blockDim.x*j + i;
		a[index] = 0;
		oA[index] = 0;
		x[index] = totalSize/n;
	}
	for(int j = n; j < n+2*ghosts; j++){
		oA[j] = 0;
	}
}

__device__ void setA(int x, double init, double* a){
	a[x] = init;
}

__device__ double linInterp(double* in){	//dangerous function, need to make sure you're only using it on the in-bounds parts of array
	return ((*(in+1) + *in)/2) - ((*in + *(in-1))/2);
}

__device__ double colellaEvenInterp(double* in){
	double aRj = 7.0f/12*(*in + *(in+1)) - 1.0f/12*(*(in+2) + *(in-1));
	if((aRj > *in && aRj < *(in+1)) || (aRj < *in && aRj > *(in+1)))
		return aRj;
	else return *in;
}

__device__ double colellaEvenInterp(double ai, double air1, double ail1, double air2, double ail2){
	return (7.0/12)*(air1-ail1) - (1.0/12)*(air2 + ail1) - (air1 + ail2);
}


__global__ void advect(double* a, double* oA, double* x, double* prev, double u, int n, int ghosts, double* minDx, double* dt, double* timeElapsed, int* counter, double* error, double tmax){
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int l_n = n;
	double l_dt, l_tElapsed = 0, l_dx = x[0], l_u = u, l_tmax = tmax;
	int l_ghosts = ghosts;
	int step = 0;
	const double courantFactor = 0.01;
	l_dt = (l_dx/l_u) * courantFactor;
	__syncthreads();
	const int maxStep = 99999;


	while(l_tElapsed < l_tmax && step < maxStep){
		__syncthreads();
		if(i == 0)
			if(step % 1000 == 0 || l_tElapsed + l_dt > l_tmax)
				for(int j = 0; j < l_n; j++){
					printf("%d %d %f %f\n", j, step, l_tElapsed, a[j]);
				}
		__syncthreads();
		for(int j = threadIdx.x; j < l_n; j+=blockDim.x){
			prev[j+l_ghosts] = a[j];
		}
		__syncthreads();
		if(i == 0){
			for(int j = 0; j < l_ghosts; j++){
				prev[j] = prev[l_ghosts];
				prev[j + l_n + l_ghosts] = prev[l_n + l_ghosts - 1];
			}
		}
		__syncthreads();
		// if(i == 0){
		// 	for(int j = 0; j < l_n + 2*l_ghosts; j++){
		// 		printf("prev %d %d %f\n", j, step, prev[j]);
		// 	}
		// 	// for(int j = 0; j < l_n; j++){
		// 	// 	printf("a %d %d %f\n", j, step, a[j]);
		// 	// }
		// }
		__syncthreads();
		for(int j = threadIdx.x; j < l_n; j+=blockDim.x){
			// int index = j*blockDim.x+i+ghosts;

			// copy to local register for quick processing... this copy actually is less efficient than global memory refs if vars are only used once
			// double ai = a[index], xi = x[index], air1 = a[index+1], air2 = a[index+2], ail1 = a[index-1], ail2 = a[index-2];
			// a[index] = ai - l_dt*u*colellaEvenInterp(ai, air1, ail1, air2, ail2)/xi;
		
			oA[j+l_ghosts] = colellaEvenInterp(prev+j+l_ghosts);
		}

		__syncthreads();

		if(i==0){	//copy over for boundary conditions
			for(int j = 0; j < l_ghosts; j++){
				oA[j] = oA[l_ghosts];
				oA[l_n+l_ghosts+j] = oA[l_n+l_ghosts - 1];
			}
		}		
		__syncthreads();
		
		for(int j = threadIdx.x; j < l_n; j+=blockDim.x){
			a[j] = a[j] + l_u * l_dt * (oA[j-1 + l_ghosts] - oA[j + l_ghosts]) / l_dx;
		}
		__syncthreads();
		l_tElapsed += l_dt;
		step++;
	}
}

__global__ void initSinusoid(double* a, double* x, double totalX, int n, int ghosts, double shift, double amp){
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	for(int j = 0; blockDim.x*j + i < n; j++){
		int index = j*blockDim.x+i;
		double temp = 0;
		for(int z = 0; z < index; z++){
			temp += x[z];
		}
		a[index] = sinpi((temp/totalX)*2)*amp + shift;
	}
	__syncthreads();
}

__global__ void initSquare(double* a, double* x, double totalX, int n, int ghosts){
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	for(int j = 0; blockDim.x*j + i < n; j++){
		int index = j*blockDim.x+i;
		if(index > n/3 && index < 2*n/3)
			a[index] = 1.5;
		else a[index] = .5;
	}
	__syncthreads();

}


CFD::CFD(int x, double size, double uIn){
	u = uIn;
	ghosts = 3;
	dim = x;
	totalX = size;
	a = new double[dim+2*ghosts];
	numBlocks = 1;
	hipMalloc((void**)&d_a, (dim)*sizeof(double));
	hipMalloc((void**)&d_x, (dim)*sizeof(double));
	hipMalloc((void**)&d_oA, (dim+ghosts*2)*sizeof(double));
	hipDeviceSynchronize();
	initialize<<<numBlocks, 1024>>>(d_a, d_oA, d_x, totalX, dim, ghosts);
	hipDeviceSynchronize();
	// initSinusoid<<<numBlocks, 1024>>>(d_a, d_x, totalX, dim, ghosts, 1, 0.5);
	initSquare<<<numBlocks, 1024>>>(d_a, d_x, totalX, dim, ghosts);
	hipDeviceSynchronize();
}

double* CFD::getA(){
	hipDeviceSynchronize();
	hipMemcpy(a, d_a, (dim)*sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	return a;
}

int CFD::getDim(){
	return dim;
}

void CFD::setInitial(int x, double init){

}

void CFD::step(double maxtime){
	double *dt,*te, *minDx, *error, *prevA;
	int* counter;
	hipMalloc((void**)&dt, sizeof(double));
	hipMalloc((void**)&te, sizeof(double));
	hipMalloc((void**)&counter, sizeof(int));
	hipMalloc((void**)&minDx, sizeof(double));
	hipMalloc((void**)&error, sizeof(double));
	hipMalloc((void**)&prevA, sizeof(double)*dim+2*ghosts);
	hipDeviceSynchronize();
	advect<<<1, 512>>>(d_a, d_oA, d_x, prevA, u, dim, ghosts, minDx, dt, te, counter, error, maxtime);
	hipDeviceSynchronize();
}
